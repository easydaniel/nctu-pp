/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
  char tchar[20];

  /* check number of points, number of iterations */
  while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
    printf("Enter number of points along vibrating string [%d-%d]: "
        ,MINPOINTS, MAXPOINTS);
    scanf("%s", tchar);
    tpoints = atoi(tchar);
    if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
      printf("Invalid. Please enter value between %d and %d\n", 
          MINPOINTS, MAXPOINTS);
  }
  while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
    printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
    scanf("%s", tchar);
    nsteps = atoi(tchar);
    if ((nsteps < 1) || (nsteps > MAXSTEPS))
      printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
  }

  printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
  int i, j;
  float x, fac, k, tmp;

  /* Calculate initial values based on sine curve */
  fac = 2.0 * PI;
  k = 0.0; 
  tmp = tpoints - 1;
  for (j = 1; j <= tpoints; j++) {
    x = k/tmp;
    values[j] = sin (fac * x);
    k = k + 1.0;
  } 

  /* Initialize old values array */
  for (i = 1; i <= tpoints; i++) 
    oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
  float dtime, c, dx, tau, sqtau;

  dtime = 0.3;
  c = 1.0;
  dx = 1.0;
  tau = (c * dtime / dx);
  sqtau = tau * tau;
  newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__device__ inline unsigned global_thread_id() {
  /* Get global thread idx */
  return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void update_parallel(float *t_values, int nsteps, int tpoints) {
  float l_value, lo_value, ln_value;
  unsigned idx = global_thread_id();
  /* Initailize */
  lo_value = l_value = sin((2.0 * PI) * ((float)idx / (float)(tpoints - 1))); 
  for (int i = 0; i < nsteps; ++i) {
    /* Calculate Math */
    ln_value = 1.82 * l_value - lo_value;
    lo_value = l_value;
    l_value = ln_value;
  }
  if (idx == 0 || idx == tpoints - 1) {
    t_values[idx] = 0;	
  } else if (idx < tpoints - 1 && idx > 0) {
    t_values[idx] = l_value;
  }
}

void update()
{
  int i, j;

  /* Update values for each time step */
  for (i = 1; i<= nsteps; i++) {
    /* Update points along line for this time step */
    for (j = 1; j <= tpoints; j++) {
      /* global endpoints */
      if ((j == 1) || (j  == tpoints))
        newval[j] = 0.0;
      else
        do_math(j);
    }

    /* Update old values with new values */
    for (j = 1; j <= tpoints; j++) {
      oldval[j] = values[j];
      values[j] = newval[j];
    }
  }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
  int i;

  for (i = 0; i < tpoints; i++) {
    printf("%6.4f ", values[i]);
    if (i % 10 == 9)
      printf("\n");
  }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
  sscanf(argv[1],"%d",&tpoints);
  sscanf(argv[2],"%d",&nsteps);
  check_param();
  float *t_values;
  hipMalloc(&t_values, sizeof(values));
  printf("Initializing points on the line...\n");
  //init_line();
  printf("Updating all points for all time steps...\n");
  //update();
  update_parallel<<<((tpoints + 1023) >> 10), 1024>>>(t_values, nsteps, tpoints);
  hipMemcpy(values, t_values, sizeof(values), hipMemcpyDeviceToHost);
  printf("Printing final results...\n");
  printfinal();
  printf("\nDone.\n\n");

  return 0;
}
